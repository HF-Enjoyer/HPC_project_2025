#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define DX 0.15
#define BETA_STEP 0.01

#define NUM_THREADS 16

// Device functions for wavefunction calculations
__device__ double euclidean_norm(double r[3]) {
    return sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
}

__device__ double HF_wavefunction(double r1[3], double r2[3]) {
    const double pi = 3.141592653589793;
    const double Z = 2.0;
    return (pow(Z, 3)/pi) * exp(-Z*(euclidean_norm(r1) + euclidean_norm(r2)));
}

__device__ double Jastrow(double beta, double r1[3], double r2[3]) {
    double r12_vec[3] = {r1[0]-r2[0], r1[1]-r2[1], r1[2]-r2[2]};
    double r12 = euclidean_norm(r12_vec);
    return exp(r12/(2.0*(1.0 + beta*r12)));
}

__device__ double vmc_WF(double beta, double r1[3], double r2[3]) {
    return HF_wavefunction(r1, r2) * Jastrow(beta, r1, r2);
}


__device__ double dot_product(double* v1, double* v2) {
    double sum = 0.0;
    for (int i = 0; i < 3; i++) {
        sum += v1[i] * v2[i];
    }
    return sum;
}

__device__ double analytical_loc_en(double beta, double r1[3], double r2[3]){
    const double Z = 2.0;

    double r12_vec[3];
    for (int i = 0; i<3; i++) {
        r12_vec[i] = r1[i] - r2[i];
    }

    double r12 = euclidean_norm(r12_vec);
    double r1_norm = euclidean_norm(r1);
    double r2_norm = euclidean_norm(r2);

    double term1 = 1.0/r12 - Z*Z;
    double den = 1.0 + beta*r12;
    double factor = 1.0/(2.0*(den*den));
    double factor_with_dot = 1.0 - dot_product(r1, r2)/(r1_norm*r2_norm);
    double term2 = factor*((Z*(r1_norm + r2_norm)/r12)*factor_with_dot - factor - 2.0/r12 + 2.0*beta/(1+beta*r12));
    return term1 + term2;
}

__global__ void monte_carlo_kernel(
    double beta,
    double step_size,
    int n_steps,
    int n_walkers,
    int n_eq,
    double* energies,
    unsigned long seed
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n_walkers) return;

    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);  // Unique seed per thread

    double r1[3] = {0.5, 0.5, 0.5};
    double r2[3] = {-0.5, -0.5, -0.5};
    double energy_sum = 0.0;

    for (int step = 0; step < n_steps; step++) {
        int electron = (hiprand_uniform_double(&state) > 0.5) ? 1 : 2;
        double new_r[3];
        double p;

        if (electron == 1) {
            for (int j = 0; j < 3; j++) {
                new_r[j] = r1[j] + (2.0 * hiprand_uniform_double(&state) * step_size - step_size);
            }
            p = vmc_WF(beta, new_r, r2) / vmc_WF(beta, r1, r2);
        } else {
            for (int j = 0; j < 3; j++) {
                new_r[j] = r2[j] + (2.0 * hiprand_uniform_double(&state) * step_size - step_size);
            }
            p = vmc_WF(beta, r1, new_r) / vmc_WF(beta, r1, r2);
        }

        p = fmin(p * p, 1.0);
        if (hiprand_uniform_double(&state) <= p) {
            if (electron == 1) memcpy(r1, new_r, 3*sizeof(double));
            else memcpy(r2, new_r, 3*sizeof(double));
        }

        if (step >= n_eq) {
            energy_sum += analytical_loc_en(beta, r1, r2);
        }
    }

    energies[tid] = energy_sum / (n_steps - n_eq);
}

double* execute_mc(
    double* betas,
    int n_betas,
    double step_size,
    int n_walkers,
    int n_steps,
    int n_eq,
    unsigned long seed
) {
    double *answers = (double*)malloc(n_betas * sizeof(double));
    double *d_energies;

    hipMalloc(&d_energies, n_walkers * sizeof(double));

    dim3 threads(NUM_THREADS);
    dim3 blocks((n_walkers + threads.x - 1) / threads.x);

    for (int i = 0; i < n_betas; i++) {
        hipMemset(d_energies, 0, n_walkers * sizeof(double));
        
        monte_carlo_kernel<<<blocks, threads>>>(
            betas[i],
            step_size,
            n_steps,
            n_walkers,
            n_eq,
            d_energies,
            seed
        );
        hipDeviceSynchronize();

        double *h_energies = (double*)malloc(n_walkers * sizeof(double));
        hipMemcpy(h_energies, d_energies, n_walkers * sizeof(double), hipMemcpyDeviceToHost);

        double sum = 0.0;
        for (int j = 0; j < n_walkers; j++) sum += h_energies[j];
        answers[i] = sum / n_walkers;
        
        free(h_energies);
    }

    hipFree(d_energies);
    return answers;
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("Usage: %s <num_steps> <n_eq> <num_trajectories>\n", argv[0]);
        return 1;
    }

    int num_steps = atoi(argv[1]);
    int n_eq = atoi(argv[2]);
    int num_trajectories = atoi(argv[3]);
    double step_size = DX;

    int n_betas = (0.5 - 0.1)  / BETA_STEP;
    printf("N_STEPS: %d. N_EQ: %d. N_TRAJ: %d\n", num_steps, n_eq, num_trajectories);
    double betas[n_betas];
    for(int i = 0; i < n_betas; i++) {
        betas[i] = 0.1 + i * BETA_STEP;
    }

    printf("Executing MC\n");
    double* energies = execute_mc(betas, n_betas, step_size, 
                                 num_trajectories, num_steps, n_eq, time(NULL));

    FILE *fp = fopen("answers.txt", "w");
    if(!fp) {
        fprintf(stderr, "Error opening output file\n");
        return 1;
    }
    
    for(int i = 0; i < n_betas; i++) {
        fprintf(fp, "%.4f\t%.8f\n", betas[i], energies[i] * 27.2114);
    }
    fclose(fp);

    double min_energy = energies[0];
    for(int i = 1; i < n_betas; i++) {
        if(energies[i] < min_energy) {
            min_energy = energies[i];
        }
    }
    printf("Minimum energy: %.6f eV\n", min_energy * 27.2114);

    free(energies);
    return 0;
}